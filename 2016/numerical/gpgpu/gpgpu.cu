#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define XSIZE 1024
#define YSIZE 1024
#define ITERATION 100
#define NUMBER(n,i,j) (((n)&0x1? XSIZE*YSIZE :0) + (i)*YSIZE + (j))

void debug_print( float* u ){
  int i,j;
  for( i=0 ; i<XSIZE ; ++i ){
    for( j=0 ; j<YSIZE ; ++j ){
      printf("%.1e ",u[NUMBER(0,i,j)]);
    }
    printf("\n");
  }
  return ;
}

__global__ void iter(float *u,float r,int n){
  int i = blockIdx.x+1;
  int j = threadIdx.x+1;
  u[NUMBER(n+1,i,j)] 
    = (1.0-4.0*r)*u[NUMBER(n,i,j)] 
    + r*(u[NUMBER(n,i+1,j)]+u[NUMBER(n,i-1,j)]+u[NUMBER(n,i,j+1)]+u[NUMBER(n,i,j-1)]);
  return ;
}

int main(){
  int array_size = 2 * XSIZE * YSIZE * sizeof(float) ;
  float r = 0.05;
  float* u = (float*)malloc(array_size);
  int i,j,n;
  // initialize
  for( i = 0 ; i < XSIZE ; ++i ){
    for( j = 0 ; j < YSIZE ; ++j ){
      u[NUMBER(0,i,j)] = ( i==0 || i==XSIZE-1 || j==0 || j==YSIZE-1 ? 0.0 : 1.0 );
      u[NUMBER(1,i,j)] = ( i==0 || i==XSIZE-1 || j==0 || j==YSIZE-1 ? 0.0 : 1.0 );
    }
  }
  // malloc in device
  float *device_u;
  hipMalloc((void**)&device_u,array_size);
  // copy to device
  hipMemcpy(device_u,u,array_size,hipMemcpyHostToDevice);

  // get time 
  struct timeval t_begin,t_end;
  gettimeofday(&t_begin,NULL);

  for( n = 0 ; n < ITERATION ; ++n )
    iter<<<XSIZE-2,YSIZE-2>>>(device_u,r,n);
  
  // print time
  hipDeviceSynchronize();
  gettimeofday(&t_end,NULL);
  double elapsed = (double)(t_end.tv_sec-t_begin.tv_sec) + (double)(t_end.tv_usec-t_begin.tv_usec) / (1000.0*1000.0);
  printf("Elapsed time = %lf(sec)\n", elapsed );
  printf("FLOPS = %g\n" , 6.0*ITERATION*(XSIZE-2)*(YSIZE-2)/elapsed );

  // copy from device
  hipMemcpy(u,device_u,array_size,hipMemcpyDeviceToHost);
  // debug_print(u);
  return 0;
}
